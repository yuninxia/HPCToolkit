// SPDX-FileCopyrightText: Contributors to the HPCToolkit Project
//
// SPDX-License-Identifier: BSD-3-Clause

#include <hip/hip_runtime.h>
#include <iostream>

// The following code implements a "heart" or "double-diamond" recursion pattern: two recursive
// loops sharing a call edge.

__attribute__((noinline))
__device__ static void top1(volatile double* x, int i);
__attribute__((noinline))
__device__ static void top2(volatile double* x, int i);

__attribute__((noinline))
__device__ static void common2(volatile double* x, int i) {
  if (i < 1) {
    x[i] *= 2;
  } else if (i < 3) {
    top1(x, i - 1);
  } else {
    top2(x, i - 1);
  }
}

__attribute__((noinline))
__device__ static void common1(volatile double* x, int i) {
  x[i] += 1;
  common2(x, i);
}

__attribute__((noinline))
__device__ static void top1(volatile double* x, int i) {
  x[i] *= 3;
  common1(x, i);
}

__attribute__((noinline))
__device__ static void top2(volatile double* x, int i) {
  x[i] /= 3;
  common1(x, i);
}

__attribute__((noinline))
__device__ static double heart() {
  volatile double x[6] = {1, 2, 3, 4, 5, 6};
  top1(x, 5);
  top2(x, 5);
  return x[0] + x[1] + x[2] + x[3] + x[4] + x[5] + x[6];
}

// Main kernel
__global__ static void kernmain() {
  heart();
}

int main() {
  hipError_t err;

  // Check that we have a device to work on
  {
    int nDevices = 0;
    err = hipGetDeviceCount(&nDevices);
    if(err != hipSuccess || nDevices == 0) {
      std::cerr << "No devices available!\n";
      return 77;  // SKIP
    }
  }

  for(int i = 0; i < 100; i++) {
    // Launch the kernel
    kernmain<<<100, 32>>>();
    err = hipGetLastError();
    if(err != hipSuccess) {
      std::cerr << "Error during kernel launch\n";
      return 1;
    }

    // Wait for the kernel to complete
    err = hipDeviceSynchronize();
    if(err != hipSuccess) {
      std::cerr << "Error returned by kernel\n";
      return 1;
    }
  }

  return 0;
}
