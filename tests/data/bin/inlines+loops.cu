// SPDX-FileCopyrightText: Contributors to the HPCToolkit Project
//
// SPDX-License-Identifier: BSD-3-Clause


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ volatile int unconstant_i = 2;
__device__ volatile int unconstant_j = 2;
__device__ volatile int unconstant_k = 2;
__attribute__((always_inline)) inline
__device__ static void f_inlined() {
  printf("Hello, world!");
}

// 1. Standard function
__global__ void f1() {
  printf("Hello, world!");
}

// 2. Inlined function call
__global__ void f2() {
  f_inlined();
}

// 3. Loops
__global__ void f3_1() {
  for(int i = 0; i < unconstant_i; i++) {
    printf("Hello, world!");
  }
}
__global__ void f3_2() {
  for(int i = 0; i < unconstant_i; i++) {
    for(int j = 0; j < unconstant_j; j++) {
      printf("Hello, world!");
    }
  }
}
__global__ void f3_3() {
  for(int i = 0; i < unconstant_i; i++) {
    for(int j = 0; j < unconstant_j; j++) {
      for(int k = 0; k < unconstant_k; k++) {
        printf("Hello, world!");
      }
    }
  }
}

// 4. Interleaved loops + inlined calls
__attribute__((always_inline)) inline
__device__ static void f4_inline() {
  for(int k = 0; k < unconstant_k; k++) {
    printf("Hello, world!");
  }
}
__global__ void f4_1() {
  for(int i = 0; i < unconstant_i; i++) {
    f4_inline();
  }
}
__global__ void f4_2() {
  for(int i = 0; i < unconstant_i; i++) {
    for(int j = 0; j < unconstant_j; j++) {
      f4_inline();
    }
  }
}
