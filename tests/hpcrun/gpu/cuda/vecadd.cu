// SPDX-FileCopyrightText: Contributors to the HPCToolkit Project
//
// SPDX-License-Identifier: BSD-3-Clause

#include <hip/hip_runtime.h>
#include <numeric>
#include <iostream>
#include <vector>

__global__ void vectorAdd(const float* a, const float* b, float* c, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < n) {
    c[i] = a[i] + b[i] + 0.0f;
  }
}

int main() {
  hipError_t err;

  // Check that we have a device to work on
  {
    int nDevices = 0;
    err = hipGetDeviceCount(&nDevices);
    if(err != hipSuccess || nDevices == 0) {
      std::cerr << "No devices available!\n";
      return 77;  // SKIP
    }
  }

  // Allocate and initialize the host-side memory
  std::vector<float> a(5000);
  std::vector<float> b(a.size());
  std::vector<float> c(a.size());
  std::iota(a.begin(), a.end(), 1);
  std::iota(b.begin(), b.end(), 3);

  // Allocate the device-side memory
  float* d_a = nullptr;
  err = hipMalloc((void**)&d_a, a.size() * sizeof(decltype(a)::value_type));
  if(err != hipSuccess) {
    std::cerr << "Failed to allocate d_a: " << hipGetErrorString(err) << "\n";
    return 1;
  }

  float* d_b = nullptr;
  err = hipMalloc((void**)&d_b, b.size() * sizeof(decltype(b)::value_type));
  if(err != hipSuccess) {
    std::cerr << "Failed to allocate d_b: " << hipGetErrorString(err) << "\n";
    return 1;
  }

  float* d_c = nullptr;
  err = hipMalloc((void**)&d_c, c.size() * sizeof(decltype(c)::value_type));
  if(err != hipSuccess) {
    std::cerr << "Failed to allocate d_c: " << hipGetErrorString(err) << "\n";
    return 1;
  }

  // Copy the data in
  err = hipMemcpy(d_a, a.data(), a.size() * sizeof(decltype(a)::value_type), hipMemcpyHostToDevice);
  if(err != hipSuccess) {
    std::cerr << "Failed to memcpy a -> d_a: " << hipGetErrorString(err) << "\n";
    return 1;
  }

  err = hipMemcpy(d_b, b.data(), b.size() * sizeof(decltype(b)::value_type), hipMemcpyHostToDevice);
  if(err != hipSuccess) {
    std::cerr << "Failed to memcpy b -> d_b: " << hipGetErrorString(err) << "\n";
    return 1;
  }

  // Launch the kernel
  int tpb = 256;
  vectorAdd<<<(a.size() + tpb - 1) / tpb, tpb>>>(d_a, d_b, d_c, a.size());

  // Copy the result back out
  err = hipMemcpy(c.data(), d_c, c.size() * sizeof(decltype(c)::value_type), hipMemcpyDeviceToHost);
  if(err != hipSuccess) {
    std::cerr << "Failed to memcpy d_c -> c: " << hipGetErrorString(err) << "\n";
    return 1;
  }

  // Free device memory
  err = hipFree(d_a);
  if(err != hipSuccess) {
    std::cerr << "Failed to free d_a: " << hipGetErrorString(err) << "\n";
    return 1;
  }
  err = hipFree(d_b);
  if(err != hipSuccess) {
    std::cerr << "Failed to free d_b: " << hipGetErrorString(err) << "\n";
    return 1;
  }
  err = hipFree(d_c);
  if(err != hipSuccess) {
    std::cerr << "Failed to free d_c: " << hipGetErrorString(err) << "\n";
    return 1;
  }

  // Validate that the answer is correct
  for(size_t i = 0; i < c.size(); i++) {
    if(c[i] != 2*i + 4) {
      std::cerr << "Invalid result at c[" << i << "]: expected " << (i+4) << ", got " << c[i] << "\n";
      return 1;
    }
  }

  return 0;
}
